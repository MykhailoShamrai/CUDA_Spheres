#include "hip/hip_runtime.h"

#include <GLFW/glfw3.h>
#include "../includes/cuda_helper.cuh"
#include "../renderers/kernel_renderer.cuh"
#include "../renderers/cpu_renderer.cuh"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "camera_helpers.cuh"
#include <stdio.h>
#include <cuda_gl_interop.h>
#include <algorithm>
#include <string>
#include <stdexcept>

#define NUMBER_OF_SPHERES 1000
#define NUMBER_OF_LIGHTS 10
//#define WIDTH 1600
//#define HEIGHT 800
#define THREAD_NUMBER 16
#define THREAD_ROTATE_NUMBER 128

#define SENSETIVITY_OF_MOUSE 0.05f

static bool IS_ANIMATED = true;

static int old_width = 1200;
static int old_height = 600;
static int n_width = old_width;
static int n_height = old_height;

static float lastX = old_width / 2.0;
static float lastY = old_height / 2.0;
static bool dragging = false;
static bool light_rotation = false;

static float angle_y_spheres = 0.0f;
static float angle_x_spheres = 0.0f;
static float angle_y_lights = 0.0f;
static float angle_x_lights = 0.0f;

static bool gpu_render = true;

char output_text_buffer[256];

static void animation_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    if (key == GLFW_KEY_SPACE && action == GLFW_PRESS)
    {
        IS_ANIMATED = !IS_ANIMATED;
    }
    else if (key == GLFW_KEY_LEFT_SHIFT && action == GLFW_PRESS)
    {
        light_rotation = !light_rotation;
    }
    else if (key == GLFW_KEY_C && action == GLFW_PRESS)
    {
        gpu_render = !gpu_render;
    }
}

static void mouse_button_callback(GLFWwindow* window, int button, int action, int mods)
{
    if (button == GLFW_MOUSE_BUTTON_LEFT)
    {
        if (action == GLFW_PRESS)
        {
            double xpos, ypos;
            glfwGetCursorPos(window, &xpos, &ypos);
            lastX = xpos;
            lastY = ypos;
            dragging = true;
            
        }
        else if (action == GLFW_RELEASE)
        {
            dragging = false;
        }
    }
}

static void cursor_position_callback(GLFWwindow* window, double xpos, double ypos)
{
    if (dragging)
    {
        // Calculate mouse movement offsets
        float xoffset = lastX - xpos;
        float yoffset = lastY - ypos;
        lastX = xpos;
        lastY = ypos;
    
        xoffset *= SENSETIVITY_OF_MOUSE;
        yoffset *= SENSETIVITY_OF_MOUSE;

        if (light_rotation)
        {
            angle_x_lights += xoffset;
            angle_y_lights += yoffset;
        }
        else
        {
            angle_x_spheres += xoffset;
            angle_y_spheres += yoffset;
        }
    }
}

static void framebuffer_size_callback(GLFWwindow*, int new_width, int new_height)
{
    n_width = new_width;
    n_height = new_height;
    glViewport(0, 0, new_width, new_height);
}

char* get_cmd_option(char** begin, char** end, const std::string& option)
{
    char** itr = std::find(begin, end, option);
    if (itr != end && ++itr != end)
    {
        return *itr;
    }
    return 0;
}

bool cmd_option_exists(char** begin, char** end, const std::string& option)
{
    return std::find(begin, end, option) != end;
}

int main(int argc, char** argv)
{
    int num_spheres = NUMBER_OF_SPHERES;
    int num_lights = NUMBER_OF_LIGHTS;
    
    // Custom number of spheres
    if (cmd_option_exists(argv, argv + argc, "-s"))
    {
        try 
        {
            char* val = get_cmd_option(argv, argv + argc, "-s");
            int num_spheres_tmp = std::stoi(val, 0, 10);
            num_spheres = num_spheres_tmp <= 0 ? NUMBER_OF_SPHERES : num_spheres_tmp;
        }
        catch (const std::exception)
        {
            fprintf(stderr, "Wrong argument for number of spheres! Default number = %d is set\n", NUMBER_OF_SPHERES);
        }
    }
    
    if (cmd_option_exists(argv, argv + argc, "-l"))
    {
        try
        {
            char* val = get_cmd_option(argv, argv + argc, "-l"); 
            int num_lights_tmp = std::stoi(val, 0, 10);
            num_lights = num_lights_tmp <= 0 ? NUMBER_OF_SPHERES : num_lights_tmp;
        }
        catch (const std::exception)
        {
            fprintf(stderr, "Wrong argument for number of lights! Default number = %d is set\n", NUMBER_OF_LIGHTS);
        }
    }

    Spheres spheres;
    Spheres d_spheres;
    LightSources lights;
    LightSources d_lights;

    h_allocate_memory_for_spheres(&spheres, num_spheres);
    create_random_spheres(&spheres, num_spheres);

    h_allocate_memory_for_light_sources(&lights, num_lights);
    create_random_light_sources(&lights, num_lights);
   
    d_allocate_memory_for_spheres(&d_spheres, num_spheres);
    d_allocate_memory_for_light_sources(&d_lights, num_lights);

    checkCudaErrors(hipMemcpy(d_spheres.x_unrotated, spheres.x_unrotated, num_spheres * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.y_unrotated, spheres.y_unrotated, num_spheres * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.z_unrotated, spheres.z_unrotated, num_spheres * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.x, spheres.x, num_spheres * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.y, spheres.y, num_spheres * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.z, spheres.z, num_spheres * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.ka, spheres.ka, num_spheres * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.ks, spheres.ks, num_spheres * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.kd, spheres.kd, num_spheres * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.R, spheres.R, num_spheres * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.G, spheres.G, num_spheres * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.B, spheres.B, num_spheres * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.alpha, spheres.alpha, num_spheres * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.radius, spheres.radius, num_spheres * sizeof(float), hipMemcpyHostToDevice));


    checkCudaErrors(hipMemcpy(d_lights.x_unrotated, lights.x_unrotated, num_lights * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.y_unrotated, lights.y_unrotated, num_lights * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.z_unrotated, lights.z_unrotated, num_lights * sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_lights.x, lights.x, num_lights * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.y, lights.y, num_lights * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.z, lights.z, num_lights * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.R, lights.R, num_lights * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.G, lights.G, num_lights * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.B, lights.B, num_lights * sizeof(float), hipMemcpyHostToDevice));

    float3 camera_pos = make_float3(0, 0, - n_width / 2);
    float* h_bitmap = (float*)malloc(n_width * n_height * 3 * sizeof(float));
    float* d_bitmap;
    checkCudaErrors(hipMalloc((void**)&d_bitmap, n_width * n_height * 3 * sizeof(float)));


    // Create a window and a context
    glfwWindowHint(GLFW_RESIZABLE, GLFW_TRUE);
    if (!glfwInit())
        return -1;

    GLFWwindow* window = glfwCreateWindow(n_width, n_height, "CUDA SPHERES", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);
    glViewport(0, 0, n_width, n_height);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0, n_width, 0, n_height, -1, 1);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glGetString(GL_VERSION);

    glfwSetKeyCallback(window, animation_callback);
    glfwSetMouseButtonCallback(window, mouse_button_callback);
    glfwSetCursorPosCallback(window, cursor_position_callback);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

    // Initialisation of timers
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEvent_t start_mem, stop_mem;
    hipEventCreate(&start_mem);
    hipEventCreate(&stop_mem);
    hipEvent_t start_rotate, stop_rotate;
    hipEventCreate(&start_rotate);
    hipEventCreate(&stop_rotate);

    int dim_blocks_x = (n_width + THREAD_NUMBER - 1) / THREAD_NUMBER;
    int dim_blocks_y = (n_height + THREAD_NUMBER - 1) / THREAD_NUMBER;


    // block for framecounting
    int number_of_frames = 0;
    double last_time = glfwGetTime();
    double last_time_anim = glfwGetTime();
    int frame_rate = 0;

    dim3 blocks(dim_blocks_x, dim_blocks_y);
    dim3 threads(THREAD_NUMBER, THREAD_NUMBER);

    int blocks_for_rotation = (num_spheres + THREAD_ROTATE_NUMBER - 1) / THREAD_ROTATE_NUMBER;

    //float3 new_camera_pos = camera_pos;
    while (!glfwWindowShouldClose(window))
    {
       
        number_of_frames++;
        double current_time = glfwGetTime();
        if (IS_ANIMATED)
        {
            double time_diff = current_time - last_time_anim;
            angle_x_spheres += time_diff * 3.0f;
            angle_y_spheres += time_diff * 3.0f;
        }
        last_time_anim = current_time;

        if (current_time - last_time >= 1.0)
        {
            frame_rate = number_of_frames;
            number_of_frames = 0;
            last_time = current_time;
        }
        
        // Check if window was resized
        if (n_width != old_width || n_height != old_height)
        {
            free(h_bitmap);
            checkCudaErrors(hipFree(d_bitmap));
            h_bitmap = (float*)malloc(n_width * n_height * 3 * sizeof(float));
            checkCudaErrors(hipMalloc((void**)&d_bitmap, n_width * n_height * 3 * sizeof(float)));
            old_height = n_height;
            old_width = n_width;
            camera_pos = make_float3(0, 0, -n_width / 2);
            dim_blocks_x = (n_width + THREAD_NUMBER - 1) / THREAD_NUMBER;
            dim_blocks_y = (n_height + THREAD_NUMBER - 1) / THREAD_NUMBER;
            blocks = dim3(dim_blocks_x, dim_blocks_y);
        }

        angle_x_spheres = angle_x_spheres > 360.0f ? 0 : angle_x_spheres < -360.0f ? 0 : angle_x_spheres;
        angle_y_spheres = angle_y_spheres > 360.0f ? 0 : angle_y_spheres < -360.0f ? 0 : angle_y_spheres;
        angle_x_lights = angle_x_lights > 360.0f ? 0 : angle_x_lights < -360.0f ? 0 : angle_x_lights;
        angle_y_lights = angle_y_lights > 360.0f ? 0 : angle_y_lights < -360.0f ? 0 : angle_y_lights;


        float elapsed_time = 0;
        float elapsed_time_mem = 0;
        float elapsed_time_rotation = 0;
        // KERNEL PART
        if (gpu_render)
        {
            hipEventRecord(start_rotate);
            rotate_objects << <blocks_for_rotation, THREAD_ROTATE_NUMBER >> > (d_spheres, d_lights, angle_x_spheres, angle_y_spheres, angle_x_lights, angle_y_lights,
                num_spheres, num_lights);
            checkCudaErrors(hipGetLastError());
            checkCudaErrors(hipDeviceSynchronize());
            hipEventRecord(stop_rotate);
            hipEventRecord(start);
            
            unsigned shmem_size = sizeof(unsigned char) * num_spheres;
            refresh_bitmap << <blocks, threads, shmem_size >> > (d_bitmap, d_spheres, num_spheres, d_lights, num_lights,
                n_width, n_height, camera_pos);
            checkCudaErrors(hipGetLastError());
            checkCudaErrors(hipDeviceSynchronize());
            hipEventRecord(stop);
            
            hipEventRecord(start_mem);
            checkCudaErrors(hipMemcpy(h_bitmap, d_bitmap, n_width * n_height * 3 * sizeof(float), hipMemcpyDeviceToHost));
            hipEventRecord(stop_mem);
            hipEventElapsedTime(&elapsed_time, start, stop);
            hipEventElapsedTime(&elapsed_time_mem, start_mem, stop_mem);
            hipEventElapsedTime(&elapsed_time_rotation, start_rotate, stop_rotate);
           
        }
        else
        {
            double start_rot = glfwGetTime();
            for (int i = 0; i < num_spheres; i++)
            {
                rotate_positions(&spheres.x[i], &spheres.z[i], &spheres.x_unrotated[i], &spheres.z_unrotated[i], angle_x_spheres);
                rotate_positions(&spheres.y[i], &spheres.z[i], &spheres.y_unrotated[i], &spheres.z[i], angle_y_spheres);
            }
            for (int i = 0; i < num_lights; i++)
            {
                rotate_positions(&lights.x[i], &lights.z[i], &lights.x_unrotated[i], &lights.z_unrotated[i], angle_x_lights);
                rotate_positions(&lights.y[i], &lights.z[i], &lights.y_unrotated[i], &lights.z[i], angle_y_lights);
            }
            double end_rot = glfwGetTime();
            elapsed_time_rotation = (end_rot - start_rot) * 1000;
            double start_render = glfwGetTime();
            refresh_bitmap_cpu(h_bitmap, spheres, num_spheres, lights, num_lights, n_width,
                n_height, camera_pos);
            double end_render = glfwGetTime();
            elapsed_time = (end_render - start_render) * 1000;
        }
        
        sprintf(output_text_buffer, "FPS: %d :: MEMORY COPY %.4f :: FRAME GENERATION FUNCTION :: %.4f :: ROTATION %.4f", frame_rate, elapsed_time_mem, elapsed_time, elapsed_time_rotation);
        
        glfwSetWindowTitle(window, output_text_buffer);
        glClear(GL_COLOR_BUFFER_BIT);

        
        glDrawPixels(n_width, n_height, GL_RGB, GL_FLOAT, h_bitmap);
        glfwSwapBuffers(window);

        glfwPollEvents();

    }
    glfwTerminate();
   
    // cleaning 
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(start_mem);
    hipEventDestroy(stop_mem);

    free(h_bitmap);
    checkCudaErrors(hipFree(d_bitmap));
    d_clean_memory_for_spheres(&d_spheres);
    h_clean_memory_for_light_sources(&lights);
    d_clean_memory_for_light_sources(&d_lights);
    h_clean_memory_for_spheres(&spheres);
    return 0;
}